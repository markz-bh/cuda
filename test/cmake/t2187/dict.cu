#include "dict.cuh"
#include <hip/hip_runtime_api.h>

__device__ 
Dict::Dict(int max_dict_size, int max_array_size) : max_size(max_dict_size) {
	data = new KeyValuePair[max_size];
	for (int i = 0; i < max_size; ++i) {
		data[i].value = new float[max_array_size];
		data[i].value_size = max_array_size;
	}
}

__device__
Dict::~Dict() {
	for (int i = 0; i < max_size; i++) { delete[] data[i].value; }
	delete[] data;
}

__device__
void Dict::add_entry(float key, float value[], int position) {
	if (position == max_size) { return; }

	data[position].key = key;
	int value_size = data[position].value_size;
	for (int i = 0; i < value_size; i++) {
		data[position].value[i] = value[i];
	}
}


__device__
void Dict::get_value(float key, float value[]) {
	for (int i = 0; i < max_size; i++) {
		if (data[i].key == key) {
			for (int j = 0; j < data[i].value_size; j++) { value[j] = data[i].value[j]; }
		}
	}
}