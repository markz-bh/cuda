#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <>
#include "dict.cuh"
using namespace std;

/*
__device__   
void cuda_max(float i, float j, float& result) {
	if (i > j) { result = i; } result = j;
}
__device__  
void  cuda_min(float i, float j, float& result) {
	if (i > j) { result = j; } result = i;
}
*/
__global__  
void Set_on_Gpu(int max_dict_size, int max_array_size, Dict** d_dict) {
	int thr_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thr_idx == 0) { *d_dict = new Dict(max_dict_size, max_array_size); }
}

__global__ 
void Add_on_Gpu(float key, float value[], int position, Dict** d_dict) {
	int thr_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thr_idx == 0) { (*d_dict)->add_entry(key, value, position); }
}

__global__  
void Show_on_Gpu(float key, Dict** d_dict, int size, float* res) {
	int thr_idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (thr_idx == 0) {
		(*d_dict)->get_value(key, res);
	}
}

int main() {
	int max_array_size = 3;
	int max_dict_size = 1;
	float key = 0.;

	float* values = (float*)malloc(3 * sizeof(float));
	float* h_result = (float*)malloc(3 * sizeof(float));
	values[0] = 1.0;
	values[1] = 2.0;
	values[2] = 3.0;

	float* d_values;
	float* d_result;

	hipMalloc((void**)&d_values, 3 * sizeof(float));
	hipMemcpy(d_values, values, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&d_result, 3 * sizeof(float));

	Dict** dev_test;
	hipMalloc((void**)&dev_test, sizeof(Dict*));

	Set_on_Gpu<<<32, 64 >>>(max_dict_size, max_array_size, dev_test);
	Add_on_Gpu<<<32, 64 >>>(key, d_values, 0, dev_test);
	Show_on_Gpu << <32, 64 >> > (key, dev_test, max_array_size, d_result);

	hipMemcpy(h_result, d_result, 3 * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < 3; i++)
		cout << h_result[i] << endl;
	return 0;
}